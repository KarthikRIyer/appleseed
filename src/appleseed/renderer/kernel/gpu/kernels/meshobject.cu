#include "hip/hip_runtime.h"

//
// This source file is part of appleseed.
// Visit http://appleseedhq.net/ for additional information and resources.
//
// This software is released under the MIT license.
//
// Copyright (c) 2018 Esteban Tovagliari, The appleseedhq Organization
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//

// appleseed.renderer headers.
#include "renderer/kernel/gpu/kernels/shadingpoint.cuh"

// OptiX headers.
#include <optix_world.h>

using namespace optix;
using namespace foundation;
using namespace renderer::gpu;

//
// Inputs.
//

rtBuffer<float3> vertices;
rtBuffer<int3>   face_indices;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


//
// Attributes.
//

rtDeclareVariable(ShadingPoint, shading_point_attr, attribute shading_point, );


//
// Intersect.
//

RT_PROGRAM void intersect(int primitive_index)
{
    // Fetch triangle vertices.
    const int3 indices = face_indices[primitive_index];
    const float3 p0 = vertices[indices.x];
    const float3 p1 = vertices[indices.y];
    const float3 p2 = vertices[indices.z];

    // Intersect ray with triangle.
    float3 n;
    float t, beta, gamma;

    if (intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma))
    {
        if (rtPotentialIntersection(t))
        {
            ShadingPoint shading_point;
            shading_point.m_flags = 1;
            shading_point.m_distance = t;
            //shading_point.geometric_normal = normalize(n);
            // ...

            shading_point_attr = shading_point;
            rtReportIntersection(0);
        }
    }
}


//
// Bounds.
//

RT_PROGRAM void bounds(int primitive_index, float result[6])
{
    const int3 indices = face_indices[primitive_index];

    const float3 v0 = vertices[indices.x];
    const float3 v1 = vertices[indices.y];
    const float3 v2 = vertices[indices.z];
    const float  area = length(cross(v1 - v0, v2 - v0));

    optix::Aabb* aabb = (optix::Aabb*)result;

    if(area > 0.0f && !isinf(area))
    {
        aabb->m_min = fminf(fminf(v0, v1), v2);
        aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
    }
    else
        aabb->invalidate();
}

