#include "hip/hip_runtime.h"

//
// This source file is part of appleseed.
// Visit http://appleseedhq.net/ for additional information and resources.
//
// This software is released under the MIT license.
//
// Copyright (c) 2018 Esteban Tovagliari, The appleseedhq Organization
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//

// appleseed.renderer headers.

// appleseed.renderer headers.
#include "renderer/kernel/gpu/kernels/ray.cuh"
#include "renderer/kernel/gpu/kernels/shadingpoint.cuh"

// OptiX headers.
#include <optix_world.h>

using namespace renderer;
using namespace foundation;
using namespace optix;


//
// Launch variables.
//

rtDeclareVariable(unsigned int, launch_index, rtLaunchIndex, );


//
// Scene variables.
//

rtDeclareVariable(rtObject, scene, , );


//
// Ray buffer.
//

rtBuffer<gpu::Ray, 1> rays;


//
// Outputs.
//

rtBuffer<gpu::ShadingPoint, 1> shading_points;


//
// Intersection programs.
//

RT_PROGRAM void raycast()
{
    gpu::ShadingPoint ray_shading_point;
    ray_shading_point.m_flags = 0;
    // todo: init ray shading point here...

    gpu::Ray ray = rays[launch_index];

    rtTrace(
        scene,
        optix::make_Ray(
            make_float3(ray.m_org.x, ray.m_org.y, ray.m_org.z),
            make_float3(ray.m_dir.x, ray.m_dir.y, ray.m_dir.z),
            0, // OptiX ray type.
            ray.m_tmin,
            ray.m_tmax),
            ray_shading_point);

    // Copy ray payload to output buffer.
    shading_points[launch_index] = ray_shading_point;
}
